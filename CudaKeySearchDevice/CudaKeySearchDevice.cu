#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include "KeySearchTypes.h"
#include "CudaKeySearchDevice.h"
#include "ptx.cuh"
#include "secp256k1.cuh"

#include "sha256.cuh"
#include "ripemd160.cuh"

#include "secp256k1.h"

#include "CudaHashLookup.cuh"
#include "CudaAtomicList.cuh"
#include "CudaDeviceKeys.cuh"

__constant__ unsigned int _INC_X[8];

__constant__ unsigned int _INC_Y[8];

__constant__ unsigned int *_CHAIN[1];

__constant__ unsigned int _TARGET_PUBKEY[8] = { 0xb4a72e4a, 0xaa69ba04, 0xb80c6891, 0xdf01f50d, 0x191a65ec, 0xcc61e4e9, 0x862d1e42, 0x1ce815b3 };
//__constant__ unsigned int _TARGET_PUBKEY[8] = { 0xdc6c9273, 0x4f925f9f, 0x88607afa, 0x26184554, 0x0c0768ab, 0x20dd5bc8, 0xcf9b43aa, 0x6bce911c };

static unsigned int *_chainBufferPtr = NULL;


__device__ void doRMD160FinalRound(const unsigned int hIn[5], unsigned int hOut[5])
{
    const unsigned int iv[5] = {
        0x67452301,
        0xefcdab89,
        0x98badcfe,
        0x10325476,
        0xc3d2e1f0
    };

    for(int i = 0; i < 5; i++) {
        hOut[i] = endian(hIn[i] + iv[(i + 1) % 5]);
    }
}


/**
 * Allocates device memory for storing the multiplication chain used in
 the batch inversion operation
 */
hipError_t allocateChainBuf(unsigned int count)
{
    hipError_t err = hipMalloc(&_chainBufferPtr, count * sizeof(unsigned int) * 8);

    if(err) {
        return err;
    }

    err = hipMemcpyToSymbol(HIP_SYMBOL(_CHAIN), &_chainBufferPtr, sizeof(unsigned int *));
    if(err) {
        hipFree(_chainBufferPtr);
    }

    return err;
}

void cleanupChainBuf()
{
    if(_chainBufferPtr != NULL) {
        hipFree(_chainBufferPtr);
        _chainBufferPtr = NULL;
    }
}

/**
 *Sets the EC point which all points will be incremented by
 */
hipError_t setIncrementorPoint(const secp256k1::uint256 &x, const secp256k1::uint256 &y)
{
    unsigned int xWords[8];
    unsigned int yWords[8];

    x.exportWords(xWords, 8, secp256k1::uint256::BigEndian);
    y.exportWords(yWords, 8, secp256k1::uint256::BigEndian);

    hipError_t err = hipMemcpyToSymbol(HIP_SYMBOL(_INC_X), xWords, sizeof(unsigned int) * 8);
    if(err) {
        return err;
    }

    return hipMemcpyToSymbol(HIP_SYMBOL(_INC_Y), yWords, sizeof(unsigned int) * 8);
}



__device__ void hashPublicKey(const unsigned int *x, const unsigned int *y, unsigned int *digestOut)
{
    unsigned int hash[8];

    sha256PublicKey(x, y, hash);

    // Swap to little-endian
    for(int i = 0; i < 8; i++) {
        hash[i] = endian(hash[i]);
    }

    ripemd160sha256NoFinal(hash, digestOut);
}

__device__ void hashPublicKeyCompressed(const unsigned int *x, unsigned int yParity, unsigned int *digestOut)
{
    unsigned int hash[8];

    sha256PublicKeyCompressed(x, yParity, hash);

    // Swap to little-endian
    for(int i = 0; i < 8; i++) {
        hash[i] = endian(hash[i]);
    }

    ripemd160sha256NoFinal(hash, digestOut);
}


__device__ void setResultFound(int idx, bool findAddress, bool compressed, unsigned int x[8], unsigned int y[8], unsigned int digest[5])
{
    CudaDeviceResult r;

    r.block = blockIdx.x;
    r.thread = threadIdx.x;
    r.idx = idx;
    r.compressed = compressed;

    for(int i = 0; i < 8; i++) {
        r.x[i] = x[i];
        r.y[i] = y[i];
    }

    doRMD160FinalRound(digest, r.digest);

    atomicListAdd(&r, sizeof(r));
}

__device__ void doIteration(int pointsPerThread, int compression, bool findAddress)
{
    unsigned int *chain = _CHAIN[0];
    unsigned int *xPtr = ec::getXPtr();
    unsigned int *yPtr = ec::getYPtr();

    // Multiply together all (_Gx - x) and then invert
    unsigned int inverse[8] = {0,0,0,0,0,0,0,1};
    for(int i = 0; i < pointsPerThread; i++) {
        unsigned int x[8];

        readInt(xPtr, i, x);

        if (findAddress) {
            unsigned int digest[5];
            if (compression == PointCompressionType::UNCOMPRESSED || compression == PointCompressionType::BOTH) {
                unsigned int y[8];
                readInt(yPtr, i, y);

                hashPublicKey(x, y, digest);

                if (checkHash(digest)) {
                    setResultFound(i, true, false, x, y, digest);
                }
            }

            if (compression == PointCompressionType::COMPRESSED || compression == PointCompressionType::BOTH) {
                hashPublicKeyCompressed(x, readIntLSW(yPtr, i), digest);

                if (checkHash(digest)) {
                    unsigned int y[8];
                    readInt(yPtr, i, y);
                    setResultFound(i, true, true, x, y, digest);
                }
            }
        }
        else {

            bool equal = true;
            for (int i = 0; i < 8; i++) {
                equal &= (x[i] == _TARGET_PUBKEY[i]);
            }
            // todo: also check sign of y coordinate, whatever lazy
            if (equal) {
                unsigned int y[8];
                unsigned int digest[5];
                readInt(yPtr, i, y);
                hashPublicKey(x, y, digest);
                setResultFound(i, false, false, x, y, digest);
            }
        }

        beginBatchAdd(_INC_X, x, chain, i, i, inverse);
    }

    doBatchInverse(inverse);

    for(int i = pointsPerThread - 1; i >= 0; i--) {

        unsigned int newX[8];
        unsigned int newY[8];

        completeBatchAdd(_INC_X, _INC_Y, xPtr, yPtr, i, i, chain, inverse, newX, newY);

        writeInt(xPtr, i, newX);
        writeInt(yPtr, i, newY);
    }
}

__device__ void doIterationWithDouble(int pointsPerThread, int compression, bool findAddress)
{
    unsigned int *chain = _CHAIN[0];
    unsigned int *xPtr = ec::getXPtr();
    unsigned int *yPtr = ec::getYPtr();

    // Multiply together all (_Gx - x) and then invert
    unsigned int inverse[8] = {0,0,0,0,0,0,0,1};
    for(int i = 0; i < pointsPerThread; i++) {
        unsigned int x[8];


        readInt(xPtr, i, x);

        if (findAddress) {
            unsigned int digest[5];
            // uncompressed
            if (compression == PointCompressionType::UNCOMPRESSED || compression == PointCompressionType::BOTH) {
                unsigned int y[8];
                readInt(yPtr, i, y);
                hashPublicKey(x, y, digest);

                if (checkHash(digest)) {
                    setResultFound(i, true, false, x, y, digest);
                }
            }

            // compressed
            if (compression == PointCompressionType::COMPRESSED || compression == PointCompressionType::BOTH) {

                hashPublicKeyCompressed(x, readIntLSW(yPtr, i), digest);

                if (checkHash(digest)) {

                    unsigned int y[8];
                    readInt(yPtr, i, y);

                    setResultFound(i, true, true, x, y, digest);
                }
            }
        }
        else {

            bool equal = true;
            for (int j = 0; j < 8; j++) {
                equal &= (x[j] == _TARGET_PUBKEY[j]);
            }
            // todo: also check sign of y coordinate, whatever lazy
            if (equal) {
                unsigned int y[8];
                unsigned int digest[5];
                readInt(yPtr, i, y);
                hashPublicKey(x, y, digest);
                setResultFound(i, false, false, x, y, digest);
            }
        }

        beginBatchAddWithDouble(_INC_X, _INC_Y, xPtr, chain, i, i, inverse);
    }

    doBatchInverse(inverse);

    for(int i = pointsPerThread - 1; i >= 0; i--) {

        unsigned int newX[8];
        unsigned int newY[8];

        completeBatchAddWithDouble(_INC_X, _INC_Y, xPtr, yPtr, i, i, chain, inverse, newX, newY);

        writeInt(xPtr, i, newX);
        writeInt(yPtr, i, newY);
    }
}

/**
* Performs a single iteration
*/
__global__ void keyFinderKernel(int points, int compression)
{
    bool findAddress = false;
    doIteration(points, compression, findAddress);
}

__global__ void keyFinderKernelWithDouble(int points, int compression)
{
    bool findAddress = false;
    doIterationWithDouble(points, compression, findAddress);
}